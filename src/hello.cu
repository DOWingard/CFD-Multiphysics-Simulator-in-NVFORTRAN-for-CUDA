#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_kernel() {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Hello from CUDA thread %d!\n", tid);
}

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    if (err != hipSuccess || deviceCount == 0) {
        std::cout << "No CUDA devices found, skipping GPU kernel." << std::endl;
    } else {
        std::cout << "Found " << deviceCount << " CUDA device(s)" << std::endl;

        hello_kernel<<<1, 8>>>();
        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cout << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
            return 1;
        }

        std::cout << "CUDA Hello World completed!" << std::endl;
    }

    return 0;
}
